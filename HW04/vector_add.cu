#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vector_add(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1000000;  // 1 million elements
    size_t size = N * sizeof(float);

    // Allocate host memory
    float *h_A = (float*) malloc(size);
    float *h_B = (float*) malloc(size);
    float *h_C = (float*) malloc(size);

    // Initialize input vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Configure grid and block sizes
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print a sample result
    printf("C[0] = %f\n", h_C[0]);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}

