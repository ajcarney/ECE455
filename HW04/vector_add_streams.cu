#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vector_add(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

int main() {
    int N = 1000000; // 1 million elements
    size_t size = N * sizeof(float);

    // Allocate host memory
    float *A = (float*) malloc(size);
    float *B = (float*) malloc(size);
    float *C = (float*) malloc(size);

    // Initialize vectors
    for (int i = 0; i < N; i++) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Create CUDA streams
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    int half = N / 2;
    size_t half_size = size / 2;

    // Copy first half of data to device asynchronously
    hipMemcpyAsync(d_A, A, half_size, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_B, B, half_size, hipMemcpyHostToDevice, stream1);

    // Copy second half of data to device asynchronously
    hipMemcpyAsync(d_A + half, A + half, half_size, hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(d_B + half, B + half, half_size, hipMemcpyHostToDevice, stream2);

    // Kernel launch parameters
    int threads = 256;
    int blocks_half = (half + threads - 1) / threads;

    // Launch kernels on streams
    vector_add<<<blocks_half, threads, 0, stream1>>>(d_A, d_B, d_C, half);
    vector_add<<<blocks_half, threads, 0, stream2>>>(d_A + half, d_B + half, d_C + half, half);

    // Copy results back to host asynchronously
    hipMemcpyAsync(C, d_C, half_size, hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(C + half, d_C + half, half_size, hipMemcpyDeviceToHost, stream2);

    // Synchronize streams
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    // Print sample results
    printf("C[0] = %f, C[N-1] = %f\n", C[0], C[N - 1]);

    // Destroy streams
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(A);
    free(B);
    free(C);

    return 0;
}

