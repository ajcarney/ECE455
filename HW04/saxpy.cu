#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

int main() {
    int N = 1000000;  // 1 million elements
    size_t size = N * sizeof(float);

    // Allocate host memory
    float *x = (float*) malloc(size);
    float *y = (float*) malloc(size);

    // Initialize vectors
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Allocate device memory
    float *d_x, *d_y;
    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);

    // Copy data to device
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    // Configure grid and block sizes
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch SAXPY kernel
    saxpy<<<blocksPerGrid, threadsPerBlock>>>(N, 2.0f, d_x, d_y);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    // Print a sample result
    printf("y[0] = %f\n", y[0]);

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);

    // Free host memory
    free(x);
    free(y);

    return 0;
}

